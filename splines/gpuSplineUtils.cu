#include "hip/hip_runtime.h"
// MaCh3 event-by-event cross-section spline code
// Written by Richard Calland, Asher Kaboth, Clarence Wret, Kamil Skwarczynski
// 
// Contains code to run on CUDA GPUs. Essentially we load up stripped TSpline3 objects to the GPU and do the equivalent of TSpline3->Eval(double) for all events
// Now also supports TF1 evals
// Called from samplePDF/samplePDFND.cpp -> splines/SplineMonolith.cpp -> splines/gpuSplineUtils.cu

// C i/o  for printf and others
#include <stdio.h>

// CUDA specifics
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Define the macros
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError()  __cudaCheckError(__FILE__, __LINE__)

// Hard code the number of splines
// Not entirely necessary: only used for val_gpu and segment_gpu being device constants. Could move them to not being device constants
// EM: for OA2022:
// #define __N_SPLINES__ 48
// EM: for OA2024:
#define __N_SPLINES__ 200

//KS: We store coefficeints {y,b,c,d} in one array one by one, this is only to define it once rather then insert "4" all over the code
#define _nCoeff_ 4

//KS: Need it for shared memory, there is way to use dynamic shared memory but I am lazy right now
#define __BlockSize__ 1024

// CUDA_ERROR_CHECK is now defined in the makefile instead
//#define CUDA_ERROR_CHECK
//#define Weight_On_SplineBySpline_Basis

// **************************************************
//             ERROR CHECKING ROUTINES
// Also exist in hip/hip_runtime_api.h
// **************************************************

// **************************************************
// Check for a safe call on GPU
inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {
// **************************************************
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif
  return;
}

// **************************************************
// Check if there's been an error
inline void __cudaCheckError( const char *file, const int line ) {
// **************************************************
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif
  return;
}

// ******************************************
// CONSTANTS
// ******************************************

// d_NAME declares DEVICE constants (live on GPU)
__device__ __constant__ unsigned int d_n_splines;
__device__ __constant__ short int d_spline_size;
#ifndef Weight_On_SplineBySpline_Basis
__device__ __constant__ int d_n_events;
#endif
//CW: Constant memory needs to be hard-coded on compile time
// Could make this texture memory instead, but don't care enough right now...
__device__ __constant__ float val_gpu[__N_SPLINES__];
__device__ __constant__ short int segment_gpu[__N_SPLINES__];

// h_NAME declares HOST constants (live on CPU)
static short int h_spline_size  = -1;
static int h_n_params     = -1;
#ifndef Weight_On_SplineBySpline_Basis
static int h_n_events     = -1;
#endif

// ******************************************
// TEXTURES
// ******************************************
//KS: Textures are L1 cache variables which are well optimised for fetching. Make texture only for variables you often acces but rarely overwrite. There are limits on texture memory so don't use huge arrays
hipTextureObject_t text_coeff_x = 0;
#ifndef Weight_On_SplineBySpline_Basis
//KS: Map keeping track how many parmaeters applies to each event, we keep two numbers here {number of splines per event, index where splines start for a given event}
hipTextureObject_t text_nParamPerEvent = 0;
#endif


// *******************************************
//              Utils
// *******************************************

// *******************************************
//KS: Get some fancy info about VRAM usage
inline void checkGpuMem() {
// *******************************************

  float free_m, total_m,used_m;
  size_t free_t, total_t;

  hipMemGetInfo(&free_t, &total_t);

  free_m = (uint)free_t/1048576.0;
  total_m = (uint)total_t/1048576.0;
  used_m = total_m - free_m;

  printf("  Memory free %f MB, total memory %f MB, memory used %f MB\n", free_m, total_m, used_m);
}

// *******************************************
//              INITIALISE GPU
// *******************************************

// *******************************************
// Initaliser when using the x array and combined y,b,c,d array
__host__ void InitGPU_SepMany(
// *******************************************
                          float **gpu_x_array,
                          float **gpu_many_array,
                          float **gpu_weights, 

                          short int** gpu_paramNo_arr,
                          unsigned int** gpu_nKnots_arr,

                 #ifndef Weight_On_SplineBySpline_Basis
                          float **cpu_total_weights, 
                          float **gpu_total_weights, 
                          int n_events,                              
                          unsigned int** gpu_nParamPerEvent,
                  #endif   
                          unsigned int sizeof_array,
                          unsigned int n_splines,
                          int Eve_size) {

  // Allocate chunks of memory to GPU
  hipMalloc((void **) gpu_paramNo_arr, n_splines*sizeof(short int));
  CudaCheckError();

  hipMalloc((void **) gpu_nKnots_arr, n_splines*sizeof(unsigned int));
  CudaCheckError();

  hipMalloc((void **) gpu_x_array, Eve_size*sizeof(float));
  CudaCheckError();

  hipMalloc((void **) gpu_many_array, _nCoeff_*sizeof_array*sizeof(float));
  CudaCheckError();

  // Allocate memory for the array of weights to be returned to CPU
  hipMalloc((void **) gpu_weights, n_splines*sizeof(float));
  CudaCheckError();
#ifndef Weight_On_SplineBySpline_Basis
  //KS: Rather than allocate memory in standard way this fancy cuda tool allows to pin host meory which make memory trnasfer faster
  hipHostMalloc((void **) cpu_total_weights, n_events*sizeof(float));
  CudaCheckError();

  //KS: Allocate memory for the array of total weights to be returned to CPU
  hipMalloc((void **) gpu_total_weights, n_events*sizeof(float));
  CudaCheckError();
  
  //KS: Allocate memory for the map keeping track how many splines each pamreter has
  hipMalloc((void **) gpu_nParamPerEvent, 2*n_events*sizeof(unsigned int));
  CudaCheckError();
  
#endif
  
  // Print allocation info to user
  printf("  Allocated %i entries for paramNo and nKnots arrays, size = %f MB\n", n_splines, double(sizeof(short int)*n_splines+sizeof(unsigned int)*n_splines)/1.E6);
  printf("  Allocated %i entries for x coeff arrays, size = %f MB\n", Eve_size, double(sizeof(float)*Eve_size)/1.E6);
  printf("  Allocated %i entries for {ybcd} coeff arrays, size = %f MB\n", _nCoeff_*sizeof_array, double(sizeof(float)*_nCoeff_*sizeof_array)/1.E6);

  //KS: Ask CUDA about memory usage
  checkGpuMem();
}

// *******************************************
// Initaliser when using the x array and combined y,b,c,d array
__host__ void InitGPU_TF1(
// *******************************************
                          float **gpu_coeffs,
                          short int** gpu_paramNo_arr,
                          short int** gpu_nPoints_arr,
                          float **gpu_weights, 
                             
                    #ifndef Weight_On_SplineBySpline_Basis
                          float **cpu_total_weights, 
                          float **gpu_total_weights, 
                          int n_events,
                              
                          unsigned int** gpu_nParamPerEvent,
                    #endif  
                          unsigned int n_splines) {

  // Holds the parameter number
  hipMalloc((void **) gpu_paramNo_arr, n_splines*sizeof(short int));
  CudaCheckError();

  // Holds the number of points
  hipMalloc((void **) gpu_nPoints_arr, n_splines*sizeof(short int));
  CudaCheckError();

  // Holds the coefficients (5th order polynomial and constant term == 1) -> 5
  hipMalloc((void **) gpu_coeffs, 5*n_splines*sizeof(float));
  CudaCheckError();

  // Allocate memory for the array of weights to be returned to CPU
  hipMalloc((void **) gpu_weights, n_splines*sizeof(float));
  CudaCheckError();

#ifndef Weight_On_SplineBySpline_Basis
  //KS: Rather than allocate memory in standard way this fancy cuda tool allows to pin host meory which make memory trnasfer faster
  hipHostMalloc((void **) cpu_total_weights, n_events*sizeof(float));
  CudaCheckError();
  
  //KS: Allocate memory for the array of total weights to be returned to CPU
  hipMalloc((void **) gpu_total_weights, n_events*sizeof(float));
  CudaCheckError();
  
  //KS: Allocate memory for the map keeping track how many splines each pamreter has
  hipMalloc((void **) gpu_nParamPerEvent, 2*n_events*sizeof(unsigned int));
  CudaCheckError();
#endif
  
  // Print allocation info to user
  printf("  Allocated %i entries for paramNo and nPoints arrays, size = %f MB\n", n_splines, double(2.0*sizeof(int)*n_splines)/1.E6);
  printf("  Allocated %i entries for coefficient arrays, size = %f MB\n", 5*n_splines, double(sizeof(float)*5*n_splines)/1.E6);

  //KS: Ask CUDA about memory usage
  checkGpuMem();
}


// *******************************************
// Allocate memory for spline segments
__host__ void InitGPU_Segments(short int **segment) {
// *******************************************

  //KS: Rather than allocate memory in standard way this fancy cuda tool allows to pin host meory which make memory trnasfer faster
  hipHostMalloc((void **) segment, __N_SPLINES__*sizeof(short int));
  CudaCheckError();
}

// *******************************************
// Allocate memory for spline segments
__host__ void InitGPU_Vals(float **vals) {
// *******************************************

  //KS: Rather than allocate memory in standard way this fancy cuda tool allows to pin host meory which make memory trnasfer faster
  hipHostMalloc((void **) vals, __N_SPLINES__*sizeof(float));
  CudaCheckError();
}


// ******************************************************
//                START COPY TO GPU
// ******************************************************

// ******************************************************
// Copy to GPU for x array and separate ybcd array
__host__ void CopyToGPU_SepMany(
// ******************************************************
                            short int *gpu_paramNo_arr,
                            unsigned int *gpu_nKnots_arr,
                            float *gpu_x_array,
                            float *gpu_many_array,

                            short int *paramNo_arr,
                            unsigned int *nKnots_arr,
                            float *cpu_x_array, 
                            float *cpu_many_array, 

                    #ifndef Weight_On_SplineBySpline_Basis
                            int n_events,
                            unsigned int *cpu_nParamPerEvent,
                            unsigned int *gpu_nParamPerEvent,
                    #endif
                            int n_params, 
                            unsigned int n_splines,
                            short int spline_size,
                            unsigned int sizeof_array) {
  if (n_params != __N_SPLINES__) {
    printf("Number of splines not equal to %i, GPU code for event-by-event splines will fail\n", __N_SPLINES__);
    printf("n_params = %i\n", n_params);
    printf("%s : %i\n", __FILE__, __LINE__);
    exit(-1);
  }

  // Write to the global statics (h_* denotes host stored variable)
  h_n_params = n_params;
  h_spline_size = spline_size;
#ifndef Weight_On_SplineBySpline_Basis
  h_n_events    = n_events;
#endif
  // Copy the constants
  // Total number of valid splines for all loaded events
  hipMemcpyToSymbol(HIP_SYMBOL(d_n_splines),   &n_splines,   sizeof(n_splines));
  CudaCheckError();
  // Total spline size per spline; i.e. just the number of points or knots in the spline
  hipMemcpyToSymbol(HIP_SYMBOL(d_spline_size), &h_spline_size, sizeof(h_spline_size));
  CudaCheckError();
#ifndef Weight_On_SplineBySpline_Basis
  // Number of events
  hipMemcpyToSymbol(HIP_SYMBOL(d_n_events), &h_n_events, sizeof(h_n_events));
  CudaCheckError();
#endif
  // Copy the coefficient arrays to the GPU; this only happens once per entire Markov Chain so is OK to do multiple extensive memory copies
  hipMemcpy(gpu_many_array, cpu_many_array, sizeof(float)*sizeof_array*_nCoeff_, hipMemcpyHostToDevice);
  CudaCheckError();

  hipMemcpy(gpu_x_array, cpu_x_array, sizeof(float)*spline_size*n_params, hipMemcpyHostToDevice);
  CudaCheckError();

  //KS: Bind our texture with the GPU variable
  //KS: Tried also moving gpu_many_array to texture memory it only worked with restricted number of MC runs, most likely hit texture memory limit :(
  struct hipResourceDesc resDesc_coeff_x;
  memset(&resDesc_coeff_x, 0, sizeof(resDesc_coeff_x));
  resDesc_coeff_x.resType = hipResourceTypeLinear;
  resDesc_coeff_x.res.linear.devPtr = gpu_x_array;
  resDesc_coeff_x.res.linear.desc = hipCreateChannelDesc<float>();
  resDesc_coeff_x.res.linear.sizeInBytes = sizeof(float)*spline_size*n_params;

  // Specify texture object parameters
  struct hipTextureDesc texDesc_coeff_x;
  memset(&texDesc_coeff_x, 0, sizeof(texDesc_coeff_x));
  texDesc_coeff_x.readMode = hipReadModeElementType;

  // Create texture object
  hipCreateTextureObject(&text_coeff_x, &resDesc_coeff_x, &texDesc_coeff_x, NULL);
  CudaCheckError();

  // Also copy the parameter number for each spline onto the GPU; i.e. what spline parameter are we calculating right now
  hipMemcpy(gpu_paramNo_arr, paramNo_arr, n_splines*sizeof(short int), hipMemcpyHostToDevice);
  CudaCheckError();

  // Also copy the knot map for each spline onto the GPU;
  hipMemcpy(gpu_nKnots_arr, nKnots_arr, n_splines*sizeof(unsigned int), hipMemcpyHostToDevice);
  CudaCheckError();

  #ifndef Weight_On_SplineBySpline_Basis
  //KS: Keep track how much splines each event has  
  hipMemcpy(gpu_nParamPerEvent, cpu_nParamPerEvent, 2*n_events*sizeof(unsigned int), hipMemcpyHostToDevice);
  CudaCheckError();
  
  //KS: Bind our texture with the GPU variable
  // create a ressource descriptor based on device pointers
  struct hipResourceDesc resDesc_nParamPerEvent;
  memset(&resDesc_nParamPerEvent, 0, sizeof(resDesc_nParamPerEvent));
  resDesc_nParamPerEvent.resType = hipResourceTypeLinear;
  resDesc_nParamPerEvent.res.linear.devPtr = gpu_nParamPerEvent;
  resDesc_nParamPerEvent.res.linear.desc = hipCreateChannelDesc<unsigned int>();
  resDesc_nParamPerEvent.res.linear.sizeInBytes = 2*n_events*sizeof(unsigned int);

  // Specify texture object parameters
  struct hipTextureDesc texDesc_nParamPerEvent;
  memset(&texDesc_nParamPerEvent, 0, sizeof(texDesc_nParamPerEvent));
  texDesc_nParamPerEvent.readMode = hipReadModeElementType;

  //Finnaly create texture object
  hipCreateTextureObject(&text_nParamPerEvent, &resDesc_nParamPerEvent, &texDesc_nParamPerEvent, NULL);
  CudaCheckError();
  #endif
}

// ******************************************************
// Copy to GPU for x array and separate ybcd array
__host__ void CopyToGPU_TF1(
// ******************************************************
                            float *gpu_coeffs,
                            short int *gpu_paramNo_arr,
                            short int *gpu_nPoints_arr,

                            float *cpu_coeffs,
                            short int *paramNo_arr,
                            short int *nPoints_arr,

                  #ifndef Weight_On_SplineBySpline_Basis
                            int n_events,
                            unsigned int *cpu_nParamPerEvent,
                            unsigned int *gpu_nParamPerEvent,
                  #endif
                            int n_params,
                            unsigned int n_splines,
                            short int _max_knots) {

  if (n_params != __N_SPLINES__) {
    printf("Number of splines not equal to %i, GPU code for event-by-event splines will fail\n", __N_SPLINES__);
    printf("n_params = %i\n", n_params);
    printf("%s : %i\n", __FILE__, __LINE__);
    exit(-1);
  }

  // Write to the global statics (h_* denotes host stored variable)
  h_n_params = n_params;
  h_spline_size = _max_knots;
#ifndef Weight_On_SplineBySpline_Basis
  h_n_events    = n_events;
#endif
  // Copy the constants
  // Total number of valid splines for all loaded events
  hipMemcpyToSymbol(HIP_SYMBOL(d_n_splines),   &n_splines,   sizeof(n_splines));
  CudaCheckError();
  // Total spline size per spline; i.e. just the number of points or knots in the spline
  hipMemcpyToSymbol(HIP_SYMBOL(d_spline_size), &h_spline_size, sizeof(h_spline_size));
  CudaCheckError();

#ifndef Weight_On_SplineBySpline_Basis
  // Number of events
  hipMemcpyToSymbol(HIP_SYMBOL(d_n_events), &h_n_events, sizeof(h_n_events));
  CudaCheckError();
#endif
  // Move the coefficients
  hipMemcpy(gpu_coeffs, cpu_coeffs, n_splines*5*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();

  // Also copy the parameter number for each spline onto the GPU; i.e. what spline parameter are we calculating right now
  hipMemcpy(gpu_paramNo_arr, paramNo_arr, n_splines*sizeof(short int), hipMemcpyHostToDevice);
  CudaCheckError();

  hipMemcpy(gpu_nPoints_arr, nPoints_arr, n_splines*sizeof(short int), hipMemcpyHostToDevice);
  CudaCheckError();
  
  #ifndef Weight_On_SplineBySpline_Basis
  //KS: Keep track how much splines each event has  
  hipMemcpy(gpu_nParamPerEvent, cpu_nParamPerEvent, 2*n_events*sizeof(unsigned int), hipMemcpyHostToDevice);
  CudaCheckError();

  //KS: Bind our texture with the GPU variable
  // create a ressource descriptor based on device pointers
  struct hipResourceDesc resDesc_nParamPerEvent;
  memset(&resDesc_nParamPerEvent, 0, sizeof(resDesc_nParamPerEvent));
  resDesc_nParamPerEvent.resType = hipResourceTypeLinear;
  resDesc_nParamPerEvent.res.linear.devPtr = gpu_nParamPerEvent;
  resDesc_nParamPerEvent.res.linear.desc = hipCreateChannelDesc<float>();
  resDesc_nParamPerEvent.res.linear.sizeInBytes = 2*n_events*sizeof(unsigned int);

  // Specify texture object parameters
  struct hipTextureDesc texDesc_nParamPerEvent;
  memset(&texDesc_nParamPerEvent, 0, sizeof(texDesc_nParamPerEvent));
  texDesc_nParamPerEvent.readMode = hipReadModeElementType;

  //Lastly create texture object
  hipCreateTextureObject(&text_nParamPerEvent, &resDesc_nParamPerEvent, &texDesc_nParamPerEvent, NULL);
  CudaCheckError();
  #endif
}

// ********************************************************
//                  START GPU KERNELS
//*********************************************************
// All the GPU kernels have similar tasks but different implementations
// Essentially they perform a binary search to find which TSpline3 point is nearest to our parameter variation
// Once it knows this, we simply extract the pre-computed coefficients for that spline point and multiply together to get a weight

//*********************************************************
// Evaluate the spline on the GPU
// Using one {y,b,c,d} array
// And one {x} array
// Should be most efficient at cache hitting and memory coalescense
// But using spline segments rather than the parameter value: avoids doing binary search on GPU
__global__ void EvalOnGPU_SepMany(
    const short int* __restrict__ gpu_paramNo_arr,
    const unsigned int* __restrict__ gpu_nKnots_arr,
    const float* __restrict__ gpu_coeff_many,
    float *gpu_weights,
    const hipTextureObject_t __restrict__ text_coeff_x) {
//*********************************************************

  // points per spline is the offset to skip in the index to move between splines
  const unsigned int splineNum = (blockIdx.x * blockDim.x + threadIdx.x);

  // Note, the new arrays are arranged as:
  //       gpu_paramNo_arr has length = spln_counter (keeps track of which parameter we're using on this thread)
  //       gpu_nKnots_arrhas length = spln_counter (keeps track where current spline starts)
  //       text_coeff_x has length = n_params * spline_size
  //       gpu_coeff_many has length = nKnots * 4
  //       ...
  //       gpu_weights has length = spln_counter * spline_size

  // this is the stopping condition!
  if (splineNum < d_n_splines) {
    // This is the segment we want for this parameter variation
    // for this particular splineNum; 0 = MACCQE, 1 = pFC, 2 = EBC, etc

    //Which Parameter we are accesing
    const short int Param = gpu_paramNo_arr[splineNum];

    // Avoids doing costly binary search on GPU
    const short int segment = segment_gpu[Param];

    //KS: Segment for coeff_x is simply parmeter*max knots + segment as each parmeters has the same spacing
    const short int segment_X = Param*d_spline_size+segment;

    //KS: Find knot position in out monolitical structure
    const unsigned int CurrentKnotPos = gpu_nKnots_arr[splineNum]*_nCoeff_+segment*_nCoeff_;

    // We've read the segment straight from CPU and is saved in segment_gpu
    // polynomial parameters from the monolithic splineMonolith
    const float fY = gpu_coeff_many[CurrentKnotPos];
    const float fB = gpu_coeff_many[CurrentKnotPos+1];
    const float fC = gpu_coeff_many[CurrentKnotPos+2];
    const float fD = gpu_coeff_many[CurrentKnotPos+3];
    // The is the variation itself (needed to evaluate variation - stored spline point = dx)
    const float dx = val_gpu[Param] - tex1Dfetch<float>(text_coeff_x, segment_X);

    //CW: Wooow, let's use some fancy intrinsics and pull down the processing time by <1% from normal multiplication! HURRAY
    gpu_weights[splineNum] = fmaf(dx, fmaf(dx, fmaf(dx, fD, fC), fB), fY);
    // Or for the more "easy to read" version:
    //gpu_weights[splineNum] = (fY+dx*(fB+dx*(fC+dx*fD)));

#ifdef DEBUG
  printf("splineNum = %i/%i, paramNo = %i, variation = %f, segment = %i, fX = %f, fX+1 = %f, dx = %f, d_n_splines = %i, d_spline_size = %i, weight = %f \n", splineNum, d_n_splines, gpu_paramNo_arr[splineNum], val_gpu[Param], segment, tex1Dfetch<float>(text_coeff_x, segment_X), tex1Dfetch<float>(text_coeff_x, segment_X+1), dx, d_n_splines, d_spline_size, gpu_weights[splineNum]);
#endif
  }
}

//*********************************************************
// Evaluate the TF1 on the GPU
// Using 5th order polynomial
__global__ void EvalOnGPU_TF1( 
    const float* __restrict__ gpu_coeffs,
    const short int* __restrict__ gpu_paramNo_arr,
    const short int* __restrict__ gpu_nPoints_arr,
    float *gpu_weights) {
//*********************************************************

  // points per spline is the offset to skip in the index to move between splines
  const unsigned int splineNum = (blockIdx.x * blockDim.x + threadIdx.x);

  // Note, the new arrays are arranged as:
  //       gpu_paramNo_arr has length = spln_counter (keeps track of which parameter we're using on this thread)
  //       gpu_coeff_x has length = n_params * spline_size
  //       gpu_coeff_many has length = spln_counter * spline_size * 4
  //       ...
  //       gpu_weights has length = spln_counter * spline_size

  if (splineNum < d_n_splines) {
    // The is the variation itself (needed to evaluate variation - stored spline point = dx)
    const float x = val_gpu[gpu_paramNo_arr[splineNum]];

    // Read the coefficients
    const float a = gpu_coeffs[splineNum*d_spline_size];
    const float b = gpu_coeffs[splineNum*d_spline_size+1];
    const float c = gpu_coeffs[splineNum*d_spline_size+2];
    const float d = gpu_coeffs[splineNum*d_spline_size+3];
    const float e = gpu_coeffs[splineNum*d_spline_size+4];

    // Match these with form in SetSplines
    // Might not be great to have this if statement: maybe split two kernels?
    if (gpu_nPoints_arr[splineNum] == 5) {
      gpu_weights[splineNum] = 1 + a*x + b*x*x + c*x*x*x + d*x*x*x*x + e*x*x*x*x*x;
    } else if (gpu_nPoints_arr[splineNum] == 2) {
      gpu_weights[splineNum] = (x<=0)*(1+a*x) + (x>0)*(1+b*x);
    } else {
      printf("Big problems, I found a nPoints array which is not 5 or 2 on GPU!\n");
    }

#ifdef DEBUG
    //if (splineNum < 200) {
    if (gpu_nPoints_arr[splineNum] == 2) {
      printf("splineNum = %i, spline_size=%i, paramNo = %i, variation = %f, a = %f, b = %f, c = %f, d = %f, e = %f, weight = %f\n", splineNum, d_spline_size, gpu_paramNo_arr[splineNum], x, a, b, c, d, e, gpu_weights[splineNum] );
    }
#endif
  }
}

#ifndef Weight_On_SplineBySpline_Basis
//*********************************************************
//KS: Evaluate the total spline event weight on the GPU, as in most cases GPU is faster, even more this significant reduce memory transfer from GPU to CPU
__global__ void EvalOnGPU_TotWeight(
   const float* __restrict__ gpu_weights,
   float *gpu_total_weights,
  const hipTextureObject_t __restrict__ text_nParamPerEvent) {
//*********************************************************
  const unsigned int EventNum = (blockIdx.x * blockDim.x + threadIdx.x);
  //KS: Accesing shared memory is much much faster than global memory hence we use shared memory for calcualtion and then write to global memory
  __shared__ float shared_total_weights[__BlockSize__];
  if(EventNum < d_n_events) //stopping condition
  {
    shared_total_weights[threadIdx.x] = 1.f;
    for (unsigned int id = 0; id < tex1Dfetch<unsigned int>(text_nParamPerEvent, 2*EventNum); ++id)
    {
      shared_total_weights[threadIdx.x] *= gpu_weights[tex1Dfetch<unsigned int>(text_nParamPerEvent, 2*EventNum+1) + id];

      #ifdef DEBUG
      printf("Event = %i, Spline_Num = %i, gpu_weights = %f \n",
              EventNum, tex1Dfetch<unsigned int>(text_nParamPerEvent, 2*EventNum+1) + id, gpu_weights[tex1Dfetch<unsigned int>(text_nParamPerEvent, 2*EventNum+1) + id];
      #endif
    }
    gpu_total_weights[EventNum] = shared_total_weights[threadIdx.x];
  }
}
#endif

// *****************************************
// Run the GPU code for the separate many arrays
// As in separate {x}, {y,b,c,d} arrays
// Pass the segment and the parameter values
// (binary search already performed in samplePDFND::FindSplineSegment()
__host__ void RunGPU_SepMany(
    const short int* gpu_paramNo_arr,
    const unsigned int* gpu_nKnots_arr,

    const float *gpu_coeff_many,

    float* gpu_weights, 
#ifdef Weight_On_SplineBySpline_Basis
    float* cpu_weights,
#else
    float* gpu_total_weights,
    float* cpu_total_weights,
#endif
    // Holds the changes in parameters
    float *vals,
    // Holds the segments for parameters
    short int *segment,
    const unsigned int h_n_splines) {
// *****************************************

  dim3 block_size;
  dim3 grid_size;

  block_size.x = __BlockSize__;
  grid_size.x = (h_n_splines / block_size.x) + 1;

  // Copy the segment values to the GPU (segment_gpu), which is h_n_params long
  hipMemcpyToSymbol(HIP_SYMBOL(segment_gpu), segment, h_n_params*sizeof(short int));
  CudaCheckError();

  // Copy the parameter values values to the GPU (vals_gpu), which is h_n_params long
  hipMemcpyToSymbol(HIP_SYMBOL(val_gpu), vals, h_n_params*sizeof(float));
  CudaCheckError();

#ifdef DEBUG
  printf("\n***********************\nGPU DEBUGGING ENABLED\n***********************\n");
  printf("block_size.x = %i, grid_size.x = %i \n", block_size.x, grid_size.x);
  printf("RunGPU_SepMany segments\n");
  for (int i = 0; i < h_n_params; i++) {
    printf("val[%i] = %f in segment %i\n", i, vals[i], segment[i]);
  }
  printf("nParams = %i, n_splines = %i", h_n_params, h_n_splines);
  printf("\n***********************\nAM NOW CALLING KERNEL\n***********************\n");
#endif

  // Set the cache config to prefer L1 for the kernel
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(EvalOnGPU_SepMany), hipFuncCachePreferL1);
  EvalOnGPU_SepMany<<<grid_size, block_size>>>(
      gpu_paramNo_arr,
      gpu_nKnots_arr,

      gpu_coeff_many,

      gpu_weights,
      text_coeff_x
      );
  CudaCheckError();

#ifdef DEBUG
  printf("Evaluated kernel with SUCCESS (drink beer)\n");
#endif

//KS: We can either copy gpu_weight and calculate total weight in reweighting loop, or not copy and calcualate total weight stall at GPU, which means less memory transfer  
#ifdef Weight_On_SplineBySpline_Basis
  // Here we have to make a somewhat large GPU->CPU transfer because it's all the splines' response
  hipMemcpy(cpu_weights, gpu_weights, h_n_splines*sizeof(float), hipMemcpyDeviceToHost);
  CudaCheckError();
  #ifdef DEBUG
  printf("Copied GPU weights to CPU with SUCCESS (drink moar beer)\n");
  printf("Released calculated response from GPU with SUCCESS (drink most beer)\n");
  #endif

//KS: Else calculate Total Weight
#else
  grid_size.x = (h_n_events / block_size.x) + 1;

  #ifdef DEBUG
  printf("\n***********************\nGPU DEBUGGING ENABLED\n***********************\n");
  printf("block_size.x = %i, grid_size.x = %i \n", block_size.x, grid_size.x);
  printf("RunGPU_TotWeight\n");

  printf("nEvents = %i, n_splines = %i, d_n_params", h_n_events, h_n_splines, h_n_params);
  printf("\n***********************\nI AM NOW CALLING KERNEL\n***********************\n");
  #endif
  EvalOnGPU_TotWeight<<<grid_size, block_size>>>(
      gpu_weights,
      gpu_total_weights,
      text_nParamPerEvent
      );
  #ifdef DEBUG
  CudaCheckError();
  printf("Evaluated kernel with SUCCESS (drink tea)\n");
  #endif
  //KS: Here we have to make a somewhat large GPU->CPU transfer because it is proportional to number of events
  //KS: In the future it might be worth to calculate only weight for events which have splines, this should reduce memory transfer
  //KS: Normally code wait for memory transfer to finish before moving furhter hipMemcpyAsync means we wil continue to execute code and in a menatime keep copyin stuff.
  hipMemcpyAsync(cpu_total_weights, gpu_total_weights, h_n_events * sizeof(float), hipMemcpyDeviceToHost, 0);

  #ifdef DEBUG
  CudaCheckError();
  printf("Copied GPU total weights to CPU with SUCCESS (drink moar tea)\n");
  printf("Released calculated response from GPU with SUCCESS (drink most tea)\n");
  #endif
#endif
}

// *****************************************
// Run the GPU code for the TF1
__host__ void RunGPU_TF1(
    const float *gpu_coeffs,
    const short int* gpu_paramNo_arr,
    const short int* gpu_nPoints_arr,

    float* gpu_weights, 
#ifdef Weight_On_SplineBySpline_Basis
    float* cpu_weights,
#else
    float* gpu_total_weights,
    float* cpu_total_weights,
#endif

  // Holds the changes in parameters
    float *vals,
    const unsigned int h_n_splines) {
// *****************************************

  dim3 block_size;
  dim3 grid_size;

  block_size.x = __BlockSize__;
  grid_size.x = (h_n_splines / block_size.x) + 1;

  // Copy the parameter values values to the GPU (vals_gpu), which is h_n_params long
  hipMemcpyToSymbol(HIP_SYMBOL(val_gpu), vals, h_n_params*sizeof(float));
  CudaCheckError();

#ifdef DEBUG
  printf("\n***********************\nGPU DEBUGGING ENABLED\n***********************\n");
  printf("block_size.x = %i, grid_size.x = %i \n", block_size.x, grid_size.x);
  printf("RunGPU_TF1 segments\n");
  for (int i = 0; i < h_n_params; i++) {
    printf("val[%i] = %f \n", i, vals[i]);
  }
  printf("nParams = %i, n_splines = %i", h_n_params, h_n_splines);
  printf("\n***********************\nAM NOW CALLING KERNEL\n***********************\n");
#endif

  // Set the cache config to prefer L1 for the kernel
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(EvalOnGPU_TF1), hipFuncCachePreferL1);
  EvalOnGPU_TF1<<<grid_size, block_size>>>(
      gpu_coeffs,
      gpu_paramNo_arr,
      gpu_nPoints_arr,

      gpu_weights
      );
  CudaCheckError();

#ifdef DEBUG
  printf("Evaluated TF1 kernel with SUCCESS (drink beer)\n");
#endif
  
//KS: We can either copy gpu_weight and calculate total weight in reweighting loop, or not copy and calcualate total weight stall at GPU, which means less memory transfer  
#ifdef Weight_On_SplineBySpline_Basis
  // Here we have to make a somewhat large GPU->CPU transfer because it's all the splines' response
  hipMemcpy(cpu_weights, gpu_weights, h_n_splines*sizeof(float), hipMemcpyDeviceToHost);
  CudaCheckError();
  
  #ifdef DEBUG
  printf("Copied TF1 GPU weights to CPU with SUCCESS (drink moar beer)\n");
  printf("Released TF1 calculated response from GPU with SUCCESS (drink most beer)\n");
  #endif

//KS: Else calculate Total Weight
#else
  grid_size.x = (h_n_events / block_size.x) + 1;

  #ifdef DEBUG
  printf("\n***********************\nGPU DEBUGGING ENABLED\n***********************\n");
  printf("block_size.x = %i, grid_size.x = %i \n", block_size.x, grid_size.x);
  printf("RunGPU_TotWeight\n");

  printf("nEvents = %i, n_splines = %i, d_n_params", h_n_events, h_n_splines, h_n_params);
  printf("\n***********************\nI AM NOW CALLING KERNEL\n***********************\n");
  #endif
  EvalOnGPU_TotWeight<<<grid_size, block_size>>>(
      gpu_weights,
      gpu_total_weights,
      text_nParamPerEvent
      );
  #ifdef DEBUG
  CudaCheckError();
  printf("Evaluated kernel with SUCCESS (drink tea)\n");
  #endif
  //KS: Here we have to make a somewhat large GPU->CPU transfer because it is proportional to number of events
  //KS: In the future it might be worth to calculate only weight for events which have splines, this should reduce memory transfer
  hipMemcpy(cpu_total_weights, gpu_total_weights, h_n_events*sizeof(float), hipMemcpyDeviceToHost);
  #ifdef DEBUG
  CudaCheckError();
  printf("Copied GPU total weights to CPU with SUCCESS (drink moar tea)\n");
  printf("Released calculated response from GPU with SUCCESS (drink most tea)\n");
  #endif
#endif
}

// *****************************************
// Make sure all Cuda threads finished execution
__host__ void SynchroniseSplines() {
  hipDeviceSynchronize();
}

// *********************************
// CLEANING
// *********************************

// *********************************
// Clean up the {x},{ybcd} arrays
__host__ void CleanupGPU_SepMany( 
    short int *gpu_paramNo_arr,
    unsigned int *gpu_nKnots_arr,

    float *gpu_x_array, 
    float *gpu_many_array, 
#ifndef Weight_On_SplineBySpline_Basis
    float *gpu_total_weights,
    unsigned int *gpu_nParamPerEvent,
    float *cpu_total_weights,
#endif
    float *gpu_weights) {
// *********************************
  hipFree(gpu_paramNo_arr);
  hipFree(gpu_nKnots_arr);

  // free the coefficient arrays
  hipDestroyTextureObject(text_coeff_x);
  hipFree(gpu_x_array);
  hipFree(gpu_many_array);

  // free weights on the gpu
  hipFree(gpu_weights);
#ifndef Weight_On_SplineBySpline_Basis
  hipFree(gpu_total_weights);
  //KS: Before removing variable let's destroy texture
  hipDestroyTextureObject(text_nParamPerEvent);
  hipFree(gpu_nParamPerEvent);
  hipHostFree(cpu_total_weights);
#endif
  return;
}

// *******************************************
// Clean up pinned variables at CPU
__host__ void CleanupGPU_Segments(short int *segment, float *vals) {
// *******************************************
    hipHostFree(segment);
    hipHostFree(vals);

    return;
}

// *********************************
// Clean up the TF1 arrays
__host__ void CleanupGPU_TF1(
    float *gpu_coeffs,
    short int *gpu_paramNo_arr,
    short int *gpu_nPoints_arr,
    
#ifndef Weight_On_SplineBySpline_Basis
    float *gpu_total_weights,
    float *cpu_total_weights,
#endif
    float *gpu_weights) {
// *********************************
  hipFree(gpu_coeffs);
  hipFree(gpu_paramNo_arr);
  hipFree(gpu_nPoints_arr);
  hipFree(gpu_weights);
#ifndef Weight_On_SplineBySpline_Basis
  hipFree(gpu_total_weights);
  hipHostFree(cpu_total_weights);
#endif
  
  return;
}
